#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_math_constants.h>
#include <cmath>

#define WARP_SIZE 32
#define WARPS_PER_BLOCK 128
#define EMB_DIM 128
#define NUM_PER_THREAD 8
template <typename T>
__device__ float convert_to_float(T value) {
    // Return 0 by default, indicating misuse if not specialized correctly.
    return 0.0f;
}

template <>
__device__ float convert_to_float<c10::Half>(c10::Half value) {
    return __half2float(value);
}

template <>
__device__ float convert_to_float<float>(float value) {
    return value;
}

template <>
__device__ float convert_to_float<at::BFloat16>(at::BFloat16 value) {
    return static_cast<float>(value);
}

template <typename T>
__device__ T convert_from_float(float value) {
    // Return 0 by default, indicating misuse if not specialized correctly.
    return static_cast<T>(0);
}
template <>
__device__ uint8_t convert_from_float<uint8_t>(float value) {
    return static_cast<uint8_t>(value);
}
template <>
__device__ c10::Half convert_from_float<c10::Half>(float value) {
    return __float2half(value);
}

template <>
__device__ float convert_from_float<float>(float value) {
    return value;
}

template <>
__device__ at::BFloat16 convert_from_float<at::BFloat16>(float value) {
    return static_cast<at::BFloat16>(value);
}



template<typename T>
__global__ void quantize_with_outliers_kernel(
    uint8_t*  compressed_value,
    // 压缩好的1bit key
    T*  dequant_dst,
    // channel_wise的平均值
    T* channel_maxvalue,
    T* quant_param,
    int batch_size, int head_size, int len,int buffer_len
    ) {
    int batch_id = blockIdx.x;
    int head_id = blockIdx.y;
    int pro_id = blockIdx.z;
// batch_id和head_id不能*NUM_PER_THREAD*WARPS_PER_BLOCK 来表示位移，因为可能会有越界的线程。
    int base_index = (batch_id * head_size * len * EMB_DIM) 
                   + (head_id * len * EMB_DIM) 
                   + (pro_id * NUM_PER_THREAD*WARPS_PER_BLOCK);
    // warp排序是列主序,获取当前线程在block中的id.
    half dequant_key[8];
    size_t th_id = threadIdx.y*16+threadIdx.x;
    int sub_th_id = threadIdx.x;
    // 首先需要知道当前线程处理哪一个向量, 向量长度除每个线程处理的元素数量，得到当前处理的向量的id
    int vec_id = threadIdx.y;
    // 判断边界 numProjBlocks个block要处理len*EMBDIM个元素
    int dequant_shfit = (batch_id * head_size * buffer_len * EMB_DIM) 
                + (head_id * buffer_len * EMB_DIM) 
                + (pro_id * NUM_PER_THREAD*WARPS_PER_BLOCK) + th_id*8;
    if(pro_id * NUM_PER_THREAD*WARPS_PER_BLOCK + vec_id*128>=len*EMB_DIM){
        return;
    }
    // printf("head:%d,pro_id:%d,dequant_shfit:%d\n",cnt_head_id,cnt_proid,dequant_shfit);
    // 获取当前线程计算的偏移量，每个线程计算NUM_PER_THREAD个元素，每thread_num_per_emb个线程计算计算一个
    // 每个线程处理NUM_PER_THREAD个元素，也就是每个线程读取一个uint8_t

    uint8_t onebit_key = compressed_value[base_index/8+th_id];
    T* sub_dequant_dst = dequant_dst+dequant_shfit;
    int4* output_as_int4 = reinterpret_cast<int4*>(sub_dequant_dst);
    // 采用向量化的写入，一个int4可以存8个half
    int4 key_data;
    half* key_half_data = reinterpret_cast<half*>(&key_data);
    // 每个head有emb_dim 个量化参数，之后会选取其中的outlier_num个
    int base_index_quant_param = batch_id * head_size *EMB_DIM + head_id*EMB_DIM;
    int extra_offset = NUM_PER_THREAD*sub_th_id;
    T* channel_maxvalue_point = channel_maxvalue+base_index_quant_param+extra_offset;
    const int4* maxvalue_input_as_int4 = reinterpret_cast<const int4*>(channel_maxvalue_point);
    int4 maxvalue_data = maxvalue_input_as_int4[0];
    half* maxvalue_half_data = reinterpret_cast<half*>(&maxvalue_data);
    // printf("base_index:%d,vec_id:%d,dequant_shfit:%d\n",base_index,vec_id,dequant_shfit);
    half local_quant_param[2];
    reinterpret_cast<int*>(local_quant_param)[0] = reinterpret_cast<int*>(quant_param+ base_index/64 +vec_id*2)[0];
    float value_zp = __half2float(local_quant_param[0]);
    float value_scale = __half2float(local_quant_param[1]);
    // local_quant_param[0]=__float2half(value_zp);
    // local_quant_param[1]=__float2half(value_scale);
    #pragma unroll
    for(int i=0;i<8;++i){
        float sub_maxvalue = __half2float(maxvalue_half_data[i]);
        float dequant = ((((((onebit_key>> i)&1) ? (1) : 0)-value_zp)*value_scale))*sub_maxvalue;
        dequant_key[i] =  __float2half(dequant);
    }
    int4* true_output = reinterpret_cast<int4*>(dequant_key);
    output_as_int4[0] = true_output[0];
    return;
}


torch::TensorOptions getOptionsForType(const std::type_info& typeInfo) {
    if (typeInfo == typeid(c10::Half)) {
        return torch::TensorOptions().device(torch::kCUDA, 0).dtype(torch::kHalf);
    } else if (typeInfo == typeid(float)) {
        return torch::TensorOptions().device(torch::kCUDA, 0).dtype(torch::kFloat);
    } else if (typeInfo == typeid(at::BFloat16)) {
        return torch::TensorOptions().device(torch::kCUDA, 0).dtype(torch::kBFloat16);
    } else {
        // Default case for unexpected types
        throw std::runtime_error("Unsupported type for tensor options.");
    }
}
    // uint8_t*  compressed_value,
    // // 压缩好的1bit key
    // uint8_t*  key_outlier_quant,
    // // 压缩好的outlier
    // T*  dequant_dst,
    // // channel_wise的平均值
    // T* channel_mean,
    // T* quant_outlier_zp,
    // // outlier量化所需要的zeropoint的
    // T* quant_outlier_scale, 
    // // outlier的量化的scale
    // uint8_t* outlier_idx,
    // // outlier的索引
    // int outlier_num,
    // // outlier channel的数目
    // int batch_size, int head_size, int len
template <typename T>
torch::Tensor MyValueDeQuantCudaTemplate(
    torch::Tensor compressed_value,
    torch::Tensor dequant_dst,
    torch::Tensor channel_maxvalue,
    torch::Tensor quant_param
    ) {

    auto options = torch::TensorOptions().device(torch::kCUDA, 0).dtype(torch::kUInt8);
    auto options_outlier_norm = getOptionsForType(typeid(T));

    int batch = compressed_value.size(0);
    int head = compressed_value.size(1);
    int len = compressed_value.size(2);
    int buffer_len = dequant_dst.size(2);
    // warp_size 表示一个warp中的线程数，一般为32  WARPS_PER_BLOCK 表示一个block中warp的数量， 一个block最大的线程数是1024，因此WARPS_PER_BLOCK的最大值为32
    // 每个线程处理NUM_PER_THREAD个元素
    int numProjBlocks = (len*EMB_DIM+(NUM_PER_THREAD*WARPS_PER_BLOCK)-1) / (NUM_PER_THREAD*WARPS_PER_BLOCK);
    dim3 numBlocks(batch , head, numProjBlocks);
    dim3 threadsPerBlockDim(16,8);

    // auto key_states_ptr = key_states.data_ptr<T>();


//     Compiler hints for using L2 Persistent Cache
    // hipStream_t stream;
    // hipStreamCreate(&stream);                                                                  // Create CUDA stream
    // int device_id{0};
    // hipGetDevice(&device_id);                                                                  // Device ID


    quantize_with_outliers_kernel<<<numBlocks, threadsPerBlockDim>>>(
    compressed_value.data_ptr<uint8_t>(),
    dequant_dst.data_ptr<T>(),
    channel_maxvalue.data_ptr<T>(),
    quant_param.data_ptr<T>(),
    batch,head,len,buffer_len);
                                                         // Remove any persistent lines in L2

    return dequant_dst;
}

    // torch::Tensor compressed_value,
    // torch::Tensor key_outlier_quant,
    // torch::Tensor dequant_dst,
    // torch::Tensor channel_mean,
    // torch::Tensor outlier_idx,
    // torch::Tensor quant_outlier_zp,
    // torch::Tensor quant_outlier_scale,
    // int outlier_num
PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("My_value_dequant_half_half", &MyValueDeQuantCudaTemplate<c10::Half>, "Quantize using Half precision",
    py::arg("compressed_value"),
    py::arg("dequant_dst"),
    py::arg("channel_maxvalue"),
    py::arg("quant_param"));

}
